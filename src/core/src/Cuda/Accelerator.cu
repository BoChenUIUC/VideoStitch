#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "SyncedMemory.h"
#include "Accelerator.h"

unsigned char* cudaProjMapX;
unsigned char* cudaProjMapY;
unsigned char* cudaFinalBlendingMap;
unsigned char* cudaFrames = nullptr;
unsigned char* cudaOut = nullptr;
unsigned char* cudaSmallOut = nullptr;
unsigned char* cudaSaliency = nullptr;

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

void helloWorld() {
	printf("Hello World\n");
}

void registerRefMap(vector<unsigned char*> mapXArr, vector<unsigned char*> mapYArr, vector<unsigned char*> blendMapArr, int viewCount, int W, int H) {
	int elementCount = W * H;

	hipMalloc(&cudaProjMapX, elementCount * viewCount * sizeof(unsigned int));
	hipMalloc(&cudaProjMapY, elementCount * viewCount * sizeof(unsigned int));
	hipMalloc(&cudaFinalBlendingMap, elementCount * viewCount * sizeof(float));

	for (int v=0; v<viewCount; v++) {
		hipMemcpy(cudaProjMapX + elementCount * v * sizeof(unsigned int) / sizeof(unsigned char), mapXArr[v], elementCount * sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(cudaProjMapY + elementCount * v * sizeof(unsigned int) / sizeof(unsigned char), mapYArr[v], elementCount * sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(cudaFinalBlendingMap + elementCount * v * sizeof(float) / sizeof(unsigned char), blendMapArr[v], elementCount * sizeof(float), hipMemcpyHostToDevice);	
	}	
}

void copyFrames(vector<unsigned char*> frames, int viewCount, int channel, int fW, int fH) {
	int elementCount = fW * fH;
	if (cudaFrames == nullptr)
		hipMalloc(&cudaFrames, elementCount * viewCount * channel * sizeof(unsigned char));

	for (int v=0; v<viewCount; v++) 
		hipMemcpy(cudaFrames + elementCount * channel * v, frames[v], elementCount * channel * sizeof(unsigned char), hipMemcpyHostToDevice);	
}

__global__ void runRenderSaliencyAreaCuda(int viewCount, int vW, int vH, int vChannels, int sW, int sH, int gridSize, int renderDiameter, int cX, int cY, int oW, int oH, int oChannels, 
										unsigned char* mapX, unsigned char* mapY, unsigned char* bMap, unsigned char* frames, unsigned char* saliency, unsigned char* out, 
										int dW, int dH, unsigned char* smallOut) {
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < oW and y < oH) {
		int curO = (y * oW + x);

		// Bilinear 
		int x0 = x / gridSize;
		int y0 = y / gridSize;
		int lt = y0 * sW + x0;

		int dx = x * dW / oW;
		int dy = y * dH / oH;
		int curD = (dy * dW + dx);

		int oriR = *(smallOut + curD * oChannels);
		int oriG = *(smallOut + curD * oChannels + 1);
		int oriB = *(smallOut + curD * oChannels + 2);

		if (saliency[lt] == 0 && saliency[lt+1] == 0 && saliency[lt+sW] == 0 && saliency[lt+sW+1] == 0) {
			*(out + curO * oChannels) = oriR;
			*(out + curO * oChannels + 1) = oriG;
			*(out + curO * oChannels + 2) = oriB;
			
			return;
		}

		float s = (x % gridSize) / (float)gridSize;
		float t = (y % gridSize) / (float)gridSize;

		int sVal = (1-s) * ( (1-t) * saliency[lt] + t * saliency[lt+sW] ) + s * ( (1-t) * saliency[lt+1] + t * saliency[lt+sW+1] );

		int curO3 = curO * oChannels;
		int elementCount = oW * oH;
		int frameElementCount = vW * vH * vChannels;

		*(out + curO3) = 0;
		*(out + curO3 + 1) = 0;
		*(out + curO3 + 2) = 0;

		unsigned int* mX = (unsigned int*) mapX;
		unsigned int* mY = (unsigned int*) mapY;
		float* b = (float*) bMap;
		int tmp;

		for (int v=0; v<viewCount; v++) {
			int pX = *(mX + elementCount * v + curO);
			int pY = *(mY + elementCount * v + curO);
			if ( !(pY < 0 || pX < 0 || pX >= vW || pY >= vH) ) {
				int pOffset = (pY * vW + pX) * vChannels;

				tmp = *(out + curO3) + *(frames + frameElementCount * v + pOffset) * *(b + elementCount * v + curO);
				*(out + curO3) = tmp > 255 ? 255 : tmp;
				tmp = *(out + curO3 + 1) + *(frames + frameElementCount * v + pOffset + 1) * *(b + elementCount * v + curO + 1);
				*(out + curO3 + 1) = tmp > 255 ? 255 : tmp;
				tmp = *(out + curO3 + 2) + *(frames + frameElementCount * v + pOffset + 2) * *(b + elementCount * v + curO + 2);
				*(out + curO3 + 2) = tmp > 255 ? 255 : tmp;
			}
		}

		if (sVal <= BLEND_THRESHOLD) {
			float blendRatio = sVal / 255.f;
			// Blending
	    	tmp = (int)(blendRatio * *(out + curO3) + (1-blendRatio) * oriR);
	    	*(out + curO3) = tmp > 255 ? 255 : tmp;
	    	tmp = (int)(blendRatio * *(out + curO3 + 1) + (1-blendRatio) * oriG);
	    	*(out + curO3 + 1) = tmp > 255 ? 255 : tmp;
	    	tmp = (int)(blendRatio * *(out + curO3 + 2) + (1-blendRatio) * oriB);
	    	*(out + curO3 + 2) = tmp > 255 ? 255 : tmp;
	    }
	}
}

__global__ void runRenderSmallSizePanoCuda(int viewCount, int vW, int vH, int vChannels, int dW, int dH, int dChannels, int oW, int oH,
										unsigned char* mapX, unsigned char* mapY, unsigned char* bMap, unsigned char* frames, unsigned char* out) {
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < dW and y < dH) {
		int curD = (y * dW + x);
		int curO = ((y * oH / dH) * oW + (x * oW / dW));

		int curD3 = curD * dChannels;
		int elementCount = oW * oH ;
		int frameElementCount = vW * vH * vChannels;

		*(out + curD3) = 0;
		*(out + curD3 + 1) = 0;
		*(out + curD3 + 2) = 0;

		unsigned int* mX = (unsigned int*) mapX;
		unsigned int* mY = (unsigned int*) mapY;
		float* b = (float*) bMap;
		int tmp;

		for (int v=0; v<viewCount; v++) {
			int pX = *(mX + elementCount * v + curO);
			int pY = *(mY + elementCount * v + curO);
			if ( !(pY < 0 || pX < 0 || pX >= vW || pY >= vH) ) {
				int pOffset = (pY * vW + pX) * vChannels;

				tmp = *(out + curD3) + *(frames + frameElementCount * v + pOffset) * *(b + elementCount * v + curO);
				*(out + curD3) = tmp > 255 ? 255 : tmp;
				tmp = *(out + curD3 + 1) + *(frames + frameElementCount * v + pOffset + 1) * *(b + elementCount * v + curO + 1);
				*(out + curD3 + 1) = tmp > 255 ? 255 : tmp;
				tmp = *(out + curD3 + 2) + *(frames + frameElementCount * v + pOffset + 2) * *(b + elementCount * v + curO + 2);
				*(out + curD3 + 2) = tmp > 255 ? 255 : tmp;
			}
		}		
	}
}

void renderSaliencyAreaCuda(int viewCount, int vW, int vH, int vChannels, int sW, int sH, int gridSize, int renderDiameter, int cX, int cY, int oW, int oH, int oChannels, 
							unsigned char* blendSaliency, unsigned char* out, int dW, int dH) {
	dim3 gdim(CeilDiv(oW,32), CeilDiv(oH,16)), bdim(32,16);

	if (cudaOut == nullptr)	
		hipMalloc(&cudaOut, oW * oH * oChannels * sizeof(unsigned char));
	if (cudaSaliency == nullptr)	
		hipMalloc(&cudaSaliency, sW * sH * sizeof(unsigned char));

	// To copy the small size canvas
	//hipMemcpy(cudaOut, out, oW * oH * oChannels * sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(cudaSaliency, blendSaliency, sW * sH * sizeof(unsigned char), hipMemcpyHostToDevice);
	runRenderSaliencyAreaCuda<<<gdim, bdim>>>(viewCount, vW, vH, vChannels, sW, sH, gridSize, renderDiameter, cX, cY, oW, oH, oChannels, cudaProjMapX, cudaProjMapY, cudaFinalBlendingMap, cudaFrames, cudaSaliency, cudaOut, dW, dH, cudaSmallOut);
	
	CHECK

	hipMemcpy(out, cudaOut, oW * oH * oChannels * sizeof(unsigned char), hipMemcpyDeviceToHost);
}

void renderSmallSizePanoCuda(int viewCount, int vW, int vH, int vChannels, int dW, int dH, int dChannels, int oW, int oH, unsigned char* downOut) {
	dim3 gdim(CeilDiv(dW,32), CeilDiv(dH,16)), bdim(32,16);

	if (cudaSmallOut == nullptr)	
		hipMalloc(&cudaSmallOut, dW * dH * dChannels * sizeof(unsigned char));
	hipMemcpy(cudaSmallOut, downOut, dW * dH * dChannels * sizeof(unsigned char), hipMemcpyHostToDevice);
	runRenderSmallSizePanoCuda<<<gdim, bdim>>>(viewCount, vW, vH, vChannels, dW, dH, dChannels, oW, oH, cudaProjMapX, cudaProjMapY, cudaFinalBlendingMap, cudaFrames, cudaSmallOut);
	
	CHECK

	hipMemcpy(downOut, cudaSmallOut, dW * dH * dChannels * sizeof(unsigned char), hipMemcpyDeviceToHost);
}