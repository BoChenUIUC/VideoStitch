#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "SyncedMemory.h"
#include "Accelerator.h"

unsigned char* cudaProjMapX;
unsigned char* cudaProjMapY;
unsigned char* cudaFinalBlendingMap;
unsigned char* cudaFrames = nullptr;
unsigned char* cudaOut = nullptr;
unsigned char* cudaSaliency = nullptr;

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

void helloWorld() {
	printf("Hello World\n");
}

void registerRefMap(vector<unsigned char*> mapXArr, vector<unsigned char*> mapYArr, vector<unsigned char*> blendMapArr, int viewCount, int W, int H) {
	int elementCount = W * H;

	hipMalloc(&cudaProjMapX, elementCount * viewCount * sizeof(unsigned int));
	hipMalloc(&cudaProjMapY, elementCount * viewCount * sizeof(unsigned int));
	hipMalloc(&cudaFinalBlendingMap, elementCount * viewCount * sizeof(float));

	for (int v=0; v<viewCount; v++) {
		hipMemcpy(cudaProjMapX + elementCount * v * sizeof(unsigned int) / sizeof(unsigned char), mapXArr[v], elementCount * sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(cudaProjMapY + elementCount * v * sizeof(unsigned int) / sizeof(unsigned char), mapYArr[v], elementCount * sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(cudaFinalBlendingMap + elementCount * v * sizeof(float) / sizeof(unsigned char), blendMapArr[v], elementCount * sizeof(float), hipMemcpyHostToDevice);	
	}	
}

void copyFrames(vector<unsigned char*> frames, int viewCount, int channel, int fW, int fH) {
	int elementCount = fW * fH;
	if (cudaFrames == nullptr)
		hipMalloc(&cudaFrames, elementCount * viewCount * channel * sizeof(unsigned char));

	for (int v=0; v<viewCount; v++) 
		hipMemcpy(cudaFrames + elementCount * channel * v, frames[v], elementCount * channel * sizeof(unsigned char), hipMemcpyHostToDevice);	
}

__global__ void runRenderSaliencyAreaCuda(int viewCount, int vW, int vH, int vChannels, int sW, int sH, int gridSize, int renderDiameter, int cX, int cY, int oW, int oH, int oChannels, 
										unsigned char* mapX, unsigned char* mapY, unsigned char* bMap, unsigned char* frames, unsigned char* saliency, unsigned char* out) {
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int sX = x / gridSize;
	const int sY = y / gridSize;
	const int curS = sW * sY + sX;

	if (x < oW and y < oH and saliency[curS] > 0) {
		// For diameter
		int tmpX = cX - sX;
		tmpX = tmpX > 0 ? tmpX : tmpX * (-1);
		tmpX = tmpX < sW/2 ? tmpX : sW-tmpX;
		int tmpY = cY - sY;
		
		if ( sqrtf(tmpX*tmpX + tmpY*tmpY)*gridSize > renderDiameter)
			return;
		
		int curO = (y * oW + x);
		int curO3 = curO * oChannels;
		int elementCount = oW * oH;
		int frameElementCount = vW * vH * vChannels;

		//outImg.at<Vec3b>(y0, x0) = Vec3b(0, 0, 0);
		*(out + curO3) = 0;
		*(out + curO3 + 1) = 0;
		*(out + curO3 + 2) = 0;

		unsigned int* mX = (unsigned int*) mapX;
		unsigned int* mY = (unsigned int*) mapY;
		float* b = (float*) bMap;
		int tmp;

		for (int v=0; v<viewCount; v++) {
			int pX = *(mX + elementCount * v + curO);
			int pY = *(mY + elementCount * v + curO);
			if ( !(pY < 0 || pX < 0 || pX >= vW || pY >= vH) ) {
				int pOffset = (pY * vW + pX) * vChannels;
				tmp = *(out + curO3) + *(frames + frameElementCount * v + pOffset) * *(b + elementCount * v + curO);
				*(out + curO3) = tmp > 255 ? 255 : tmp;

				tmp = *(out + curO3 + 1) + *(frames + frameElementCount * v + pOffset + 1) * *(b + elementCount * v + curO + 1);
				*(out + curO3 + 1) = tmp > 255 ? 255 : tmp;

				tmp = *(out + curO3 + 2) + *(frames + frameElementCount * v + pOffset + 2) * *(b + elementCount * v + curO + 2);
				*(out + curO3 + 2) = tmp > 255 ? 255 : tmp;
			}
		}
	}
}

void renderSaliencyAreaCuda(int viewCount, int vW, int vH, int vChannels, int sW, int sH, int gridSize, int renderDiameter, int cX, int cY, int oW, int oH, int oChannels, 
							unsigned char* saliencyInfo, unsigned char* out) {
	dim3 gdim(CeilDiv(oW,32), CeilDiv(oH,16)), bdim(32,16);

	if (cudaOut == nullptr)	
		hipMalloc(&cudaOut, oW * oH * oChannels * sizeof(unsigned char));
	if (cudaSaliency == nullptr)	
		hipMalloc(&cudaSaliency, sW * sH * sizeof(unsigned char));

	// To copy the small size canvas
	hipMemcpy(cudaOut, out, oW * oH * oChannels * sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(cudaSaliency, saliencyInfo, sW * sH * sizeof(unsigned char), hipMemcpyHostToDevice);

	runRenderSaliencyAreaCuda<<<gdim, bdim>>>(viewCount, vW, vH, vChannels, sW, sH, gridSize, renderDiameter, cX, cY, oW, oH, oChannels, cudaProjMapX, cudaProjMapY, cudaFinalBlendingMap, cudaFrames, cudaSaliency, cudaOut);
	
	CHECK

	hipMemcpy(out, cudaOut, oW * oH * oChannels * sizeof(unsigned char), hipMemcpyDeviceToHost);
}